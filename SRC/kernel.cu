#include "hip/hip_runtime.h"


#include <stdio.h>
#include <algorithm>
#include <iterator>
#include <set>
#include <string>

#include <GL/glew.h>
#include <GL/wglew.h>
#include <GL/freeglut.h>
#include <SFML/System.hpp>
#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>


// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <>
#include <cuda_gl_interop.h>




#define NBody 16384
#define NB_THREAD 128

#define ScreenWidth 1024
#define ScreenHeight 768

#define invNBody (1.0f / NBody) //Variable de "vitesse de simulation" pour �viter une simulation explosive


__global__ void NBodyCUDA(float2 *nvBodyPos, float2 *nvBodyV, float2 *nvBodyDest, float tempInvNbody)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	//Step 3 Simuler l'interaction du body courant avec tous les autres bodies de nvBodyPos, ajouter � la vitesse, appliquer la vitesse
	
	float2 tempForce = { 0, 0 };
	
	for(int i = 0; i < NBody; i++) {

		float intGrav = 0;

		intGrav = -1.0f * (0.00001f + (nvBodyPos[index].x - nvBodyPos[i].x)*(nvBodyPos[index].x - nvBodyPos[i].x) + (nvBodyPos[index].y - nvBodyPos[i].y)*(nvBodyPos[index].y - nvBodyPos[i].y));
		tempForce.x += (nvBodyPos[index].x - nvBodyPos[i].x) / intGrav;
		tempForce.y += (nvBodyPos[index].y - nvBodyPos[i].y) / intGrav;

	}

	nvBodyV[index].x += tempForce.x * tempInvNbody;
	nvBodyV[index].y += tempForce.y * tempInvNbody;
	

	//Step 2 Copier la bonne donn�e de nvBodyPos dans nvBodyDest
	nvBodyDest[index].x = nvBodyPos[index].x + nvBodyV[index].x;
	nvBodyDest[index].y = nvBodyPos[index].y + nvBodyV[index].y;

}



int main()
{
	//Allocation des tableaux de d�part
	float2* BodyPos = (float2 *)malloc(NBody * sizeof(float2));
	float2* BodyV = (float2 *)malloc(NBody * sizeof(float2));

	//Allocation des tableaux de donn�es initiales
	float2* nvBodyPos;
	hipMalloc(&nvBodyPos, NBody * sizeof(float2));

	float2* nvBodyV;
	hipMalloc(&nvBodyV, NBody * sizeof(float2));

	float2* nvBodyDest;
	hipMalloc(&nvBodyDest, NBody * sizeof(float2));

	//Verification d'erreur
	if (nvBodyPos == NULL || nvBodyDest == NULL || nvBodyV == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	//position initiale al�atoire
	srand(time(NULL));
	for (int i = 0; i < NBody; i++)
	{
		BodyPos[i].x = (ScreenWidth / 2) + 600 * (-0.5 + (rand() / (float)RAND_MAX));
		BodyPos[i].y = (ScreenHeight / 2) + 300 * (-0.5 + (rand() / (float)RAND_MAX));
	}

	//Initialisation de SFML
	sf::VertexArray tempArray;
	tempArray.resize(NBody);
	tempArray.setPrimitiveType(sf::Points);
	sf::RenderWindow window(sf::VideoMode(1024, 768), "My window");

	int blockSize = NB_THREAD;
	int gridSize = (NBody + blockSize - 1) / blockSize;


	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event))
		{

		}
		if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) exit(0);

		//Step 1
		//Copie des positions vers le GPU
		hipMemcpy(nvBodyPos, BodyPos, NBody * sizeof(float2), hipMemcpyHostToDevice);

		//Lancement du kernel
		NBodyCUDA <<<gridSize, blockSize >>> (nvBodyPos, nvBodyV, nvBodyDest, invNBody);

		//R�cup�ration des nouvelles positions
		hipMemcpy(BodyPos, nvBodyDest, NBody * sizeof(float2), hipMemcpyDeviceToHost);

		//Copie des donn�es dans la Vertex Array pour l'affichage
		for (int i = 0; i < NBody; i++)
		{
			tempArray[i].position.x = BodyPos[i].x;
			tempArray[i].position.y = BodyPos[i].y;
		}


		//Affichage
		window.clear(sf::Color::Black);
		window.draw(tempArray);
		window.display();
		//exit(0);
	}
}

